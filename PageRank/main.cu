
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int pagerank(float *coovala,int *coorowinda,int *coocolinda,int nnz,int nodes,float q,int max_iters,float accept_key,float *pr,int *real_iters,float *real_key);

int main()
{
    FILE *fp=NULL;
    
    float *value=NULL,*pr=NULL;
    int *row=NULL,*col=NULL;
    
    float elapsedTime;
    hipEvent_t start, stop;
    
    double sum=0.0;
    float q=0.8,accept_key=0.0000001,error_value;
    int i,nnz,nodes,max_iters=100,real_iters;
    
    fp=fopen("Graph.txt","r");
    if(fp==NULL)
    {
        printf("can't read matrix.txt!\n");
        return(1);
    }
    
    printf("start load data.\n");
    
    fscanf(fp,"%d",&nodes);
    fscanf(fp,"%d",&nnz);
    
    printf("nodes=%d,nnz=%d.\nif it is not right,then please check ID_To_ID.txt,ke neng you yi xie te shu zi fu zai wen jian kai tou.\n",nodes,nnz);
    
    value=(float *)malloc(sizeof(float)*nnz);
    row=(int *)malloc(sizeof(int)*nnz);
    col=(int *)malloc(sizeof(int)*nnz);
    pr=(float *)malloc(sizeof(float)*nodes);
    
    //待定_直接使用型
    for(i=0;i<nnz;i++)
    {
        fscanf(fp,"%d",row+i);
        fscanf(fp,"%d",col+i);
        fscanf(fp,"%f",value+i);
        row[i]--;
        col[i]--;
        //test ok
        //printf("%d,%d,%f.\n",row[i],col[i],value[i]);
        //getchar();
    }
    fclose(fp);
    fp=NULL;
    
    printf("data load is ok.\nstart solve pr value.\n");
    
    //test ok
    //printf("%d,%d,%d,%d,%d.\n",row[0],row[1],row[2],row[3],row[4]);
    
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    pagerank(value,row,col,nnz,nodes,q,max_iters,accept_key,pr,&real_iters,&error_value);
    
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("pr value solve ok.\nstart write answ.\n");
    
    fp=fopen("pr_out.txt","w");
    if(fp==NULL)
    {
        printf("can't write pr_out.txt!\n");
        return(1);
    }
    fprintf(fp,"nodes:%d\n",nodes);
    fprintf(fp,"nnz:%d\n",nnz);
    fprintf(fp,"real_iters:%d\n",real_iters);
    fprintf(fp,"sigma:%f\n",error_value);
    for(i=0;i<nodes;i++)
        sum+=pr[i];
    fprintf(fp,"the sum pr value:%f\n",sum);
    fprintf(fp,"cuda_use_time_ms:%fms\n",elapsedTime);
    fclose(fp);
    fp=NULL;
    
    fp=fopen("pr_value.txt","w");
    if(fp==NULL)
    {
        printf("can't write pr_value.txt!\n");
        return(1);
    }
    for(i=0;i<nodes;i++)
        fprintf(fp,"%f\n",pr[i]*10000);
    fclose(fp);
    fp=NULL;
    
    return(0);
}
