#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hipsparse.h"

#define CLEANUP(s)                                   \
do {                                                 \
    printf ("%s\n", s);                              \
    /*if (yHostPtr)           free(yHostPtr);          \
    if (zHostPtr)           free(zHostPtr);          \
    if (xIndHostPtr)        free(xIndHostPtr);       \
    if (xValHostPtr)        free(xValHostPtr);       \
    if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);\
    if (cooColIndexHostPtr) free(cooColIndexHostPtr);\
    if (cooValHostPtr)      free(cooValHostPtr);     \
    if (y)                  hipFree(y);             \
    if (z)                  hipFree(z);             \
    if (xInd)               hipFree(xInd);          \
    if (xVal)               hipFree(xVal);          \
    if (csrRowPtr)          hipFree(csrRowPtr);     \
    if (cooRowIndex)        hipFree(cooRowIndex);   \
    if (cooColIndex)        hipFree(cooColIndex);   \
    if (cooVal)             hipFree(cooVal);        \
    if (descr)              hipsparseDestroyMatDescr(descr);\
    if (handle)             hipsparseDestroy(handle); \
    hipDeviceReset();          \
    fflush (stdout);*/                                 \
} while (0)

__global__ void assign(float *data,int length,float value)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<length)
        data[id]=value;
}

__global__ void reduction_normal(float *in,int length,float *out)
{
    __shared__ float partialSum[1024];
    int index=blockIdx.x*blockDim.x + threadIdx.x;
    int id=threadIdx.x;
    if(index<length)
        partialSum[id]=in[index];
    else
        partialSum[id]=0.0f;
    __syncthreads();

    for(unsigned int stride = blockDim.x / 2;stride > 0;stride /= 2)
    {
        __syncthreads();
        if(id < stride)
            partialSum[id]+=partialSum[id + stride];
    }
    if(threadIdx.x==0)
        out[blockIdx.x]=partialSum[0];
}

void reduction(float *data,int length,float *temp_1,float *temp_2,float *answ)
{
    float *in,*out,*temp;
    int blocksize=1024;
    
    in=data;
    out=temp_1;
    if(length<=blocksize)
        out=answ;

    reduction_normal<<<length/blocksize+1,blocksize>>>(in,length,out);
    hipDeviceSynchronize();

    if(length%blocksize==0)
        length/=blocksize;
    else
        length=length/blocksize+1;

    in=out;
    out=temp_2;

    while(length>blocksize)
    {
        reduction_normal<<<length/blocksize+1,blocksize>>>(in,length,out);
        hipDeviceSynchronize();

        if(length%blocksize==0)
            length/=blocksize;
        else
            length=length/blocksize+1;

        temp=in;
        in=out;
        out=temp;
    }

    out=answ;
    reduction_normal<<<length/blocksize+1,blocksize>>>(in,length,out);
    hipDeviceSynchronize();
}

__global__ void step_2_1(float *y,float *r,int nodes,float *answ)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<nodes)
    {
        float value_r=r[id];
        float value_y=y[id];
        answ[id]=value_r-value_y;
    }
}

void step_2(float *y,float *r,int nodes,float *temp_0,float *temp_1,float *d)
{
    step_2_1<<<nodes/256+1,256>>>(y,r,nodes,temp_1);
    hipDeviceSynchronize();
    reduction(temp_1,nodes,temp_0,temp_1,d);
}

__global__ void step_3(float *y,int nodes,float *d,float *r)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<nodes)
        r[id]=y[id]+*d/nodes;
}

__global__ void step_4_1(float *r0,float *r1,int nodes,float *r_adder_abs)
{
    int id=blockIdx.x*blockDim.x+threadIdx.x;
    if(id<nodes)
        r_adder_abs[id]=abs(r0[id]-r1[id]);
}

void step_4(float *r0,float *r1,int nodes,float *temp_0,float *temp_1,float *sigma_cpu)
{
    step_4_1<<<nodes/256+1,256>>>(r0,r1,nodes,temp_1);
    hipDeviceSynchronize();
    reduction(temp_1,nodes,temp_0,temp_1,temp_0);
    hipMemcpy(sigma_cpu,temp_0,sizeof(float),hipMemcpyDeviceToHost);
}

int pagerank(float *coovala,int *coorowinda,int *coocolinda,int nnz,int nodes,float q,int max_iters,float accept_key,float *pr,int *real_iters,float *real_key)
{
    //矩阵数据
    float *gcoovala;
    int *gcoorowinda,*gcoocolinda;

    //gpu数据
    float *r[3],*y,*sum_temp[2];
    float *d;

    //cpu端数据
    float sigma;
    int i=0;

    //申请gpu空间
    hipMalloc((void **)&gcoovala,sizeof(float)*nnz);
    hipMalloc((void **)&gcoorowinda,sizeof(int)*nnz);
    hipMalloc((void **)&gcoocolinda,sizeof(int)*nnz);
    hipMalloc((void **)&r[0],sizeof(int)*(nodes*6+1));
    r[1]=r[0]+nodes;
    r[2]=r[1]+nodes;
    y=r[2]+nodes;
    sum_temp[0]=y+nodes;
    sum_temp[1]=sum_temp[0]+nodes;
    d=sum_temp[1]+nodes;

    //复制矩阵到gpu
    hipMemcpy(gcoovala,coovala,sizeof(float)*nnz,hipMemcpyHostToDevice);
    hipMemcpy(gcoorowinda,coorowinda,sizeof(int)*nnz,hipMemcpyHostToDevice);
    hipMemcpy(gcoocolinda,coocolinda,sizeof(int)*nnz,hipMemcpyHostToDevice);
    
    //test
    /*int temp[5];
    hipMemcpy(temp,gcoorowinda,5,hipMemcpyDeviceToHost);
    printf("%d,%d,%d,%d,%d.\n",coorowinda[0],coorowinda[1],coorowinda[2],coorowinda[3],coorowinda[4]);
    printf("%d,%d,%d,%d,%d.\n",temp[0],temp[1],temp[2],temp[3],temp[4]);
    getchar();*/
    
    //转换为csr矩阵
    int *csrRowPtr;
    float beta=0;

    hipError_t cudaStat;
    hipsparseStatus_t status;
    hipsparseHandle_t handle=0;
    hipsparseMatDescr_t descr=0;
    status= hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library initialization failed");
        return 1;
    }
    status= hipsparseCreateMatDescr(&descr);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor initialization failed");
        return 1;
    }
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
    cudaStat = hipMalloc((void**)&csrRowPtr,(nodes+1)*sizeof(int));
    if (cudaStat != hipSuccess) {
        CLEANUP("Device malloc failed (csrRowPtr)");
        return 1;
    }
    status= hipsparseXcoo2csr(handle,gcoorowinda,nnz,nodes,
                             csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Conversion from COO to CSR format failed");
        return 1;
    }
    //转换完成

    //初始化数据
    assign<<<nodes/256+1,256>>>(r[0],nodes,1.0/nodes);
    hipDeviceSynchronize();
    
    while(1)
    {
        //step_1
        //利用cusparse计算
        status= hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, nodes, nodes, nnz,
            &q, descr, gcoovala, csrRowPtr, gcoocolinda,
            r[i%3],&beta, y);
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            CLEANUP("Matrix-vector multiplication failed");
            return 1;
        }
        hipDeviceSynchronize();
        
        //test
        /*hipMemcpy(pr,y,sizeof(float)*nodes,hipMemcpyDeviceToHost);
        printf("%f,%f,%f,%f,%f,%f.\n",pr[0],pr[1],pr[2],pr[3],pr[4],pr[5]);
        getchar();*/

        //step 2
        step_2(y,r[i%3],nodes,sum_temp[0],sum_temp[1],d);

        //step 3
        step_3<<<nodes/256+1,256>>>(y,nodes,d,r[(i+1)%3]);
        hipDeviceSynchronize();

        //step 4
        step_4(r[i%3],r[(i+1)%3],nodes,sum_temp[0],sum_temp[1],&sigma);
        if(sigma<accept_key||i+1>=max_iters)
            break;

        i++;
    }

    hipMemcpy(pr,r[(i+1)%3],sizeof(float)*nodes,hipMemcpyDeviceToHost);
    *real_key=sigma;
    *real_iters=i+1;

    hipFree(gcoovala);
    hipFree(gcoorowinda);
    hipFree(gcoocolinda);
    hipFree(r[0]);

    //终止使用cusparse
    hipFree(csrRowPtr);
    status = hipsparseDestroyMatDescr(descr);
    descr = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("Matrix descriptor destruction failed");
        return 1;
    }
    status = hipsparseDestroy(handle);
    handle = 0;
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        CLEANUP("CUSPARSE Library release of resources failed");
        return 1;
    }
    return(1);
}
